#include "hip/hip_runtime.h"
#include "gpuMD5.h"
#include "cutil.h"

// Forward declarations of basic MD5 functions
__device__ UINT F(UINT x, UINT y, UINT z);
__device__ UINT G(UINT x, UINT y, UINT z);
__device__ UINT H(UINT& x, UINT& y, UINT& z);
__device__ UINT I(UINT& x, UINT& y, UINT& z);

__device__ UINT ROTATE_LEFT(UINT& x, UINT& n);

__device__ void FF(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac);
__device__ void GG(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac);
__device__ void HH(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac);
__device__ void II(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac);

// F, G and H are basic MD5 functions: selection, majority, parity 
__device__ UINT F(UINT x, UINT y, UINT z){
  return ((x & y) | (~x & z));
}
__device__ UINT G(UINT x, UINT y, UINT z){
  return ((x & z) | (y & ~z));
}
__device__ UINT H(UINT& x, UINT& y, UINT& z){
  return (x ^ y ^ z);
}
__device__ UINT I(UINT& x, UINT& y, UINT& z){
  return (y ^ (x | ~z));
}

// ROTATE_LEFT rotates x left n bits 
__device__ UINT ROTATE_LEFT(UINT& x, UINT& n){
  return ((x << n) | (x >> (32 - n)));
}

// FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4
// Rotation is separate from addition to prevent recomputation 
__device__ void FF(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac){
  a += F(b, c, d) + x + ac; 
  a =  ROTATE_LEFT(a, s); 
  a += b;
}
__device__ void GG(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac){
  a += G(b, c, d) + x + ac; 
  a =  ROTATE_LEFT(a, s); 
  a += b;
}
__device__ void HH(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac){  
  (a) += H ((b), (c), (d)) + (x) + (UINT)(ac); 
   (a) = ROTATE_LEFT ((a), (s)); 
   (a) += (b); 
}
__device__ void II(UINT& a, UINT& b, UINT& c, UINT& d, UINT& x, UINT s, UINT ac){ 
(a) += I ((b), (c), (d)) + (x) + (UINT)(ac); 
   (a) = ROTATE_LEFT ((a), (s)); 
   (a) += (b);
}
// CONSTANT DECLARATIONS
extern __shared__ char array[];
//__constant__ int device_shift_amounts[64];
//__constant__ UINT device_sines[64];
__constant__ UINT deviceTarget[4];
__device__ int resultIndex;

__global__ void md5Hash(UCHAR**, int*, uint4*);
__device__ UINT* pad(UCHAR*, int);

void initialiseConstants(UINT* target) {
  UINT nf = -1;
/*
  int host_shift_amounts[] = {7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,
             5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,
             4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,
             6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21};
  hipMemcpyToSymbol(HIP_SYMBOL(device_shift_amounts), host_shift_amounts, sizeof(host_shift_amounts));

  UINT host_sines[] = {3614090360, 3905402710, 606105819, 3250441966, 4118548399, 1200080426, 2821735955,
                       4249261313, 1770035416, 2336552879, 4294925233, 2304563134, 1804603682, 4254626195, 
                       2792965006, 1236535329, 4129170786, 3225465664, 643717713, 3921069994, 3593408605,
                       38016083, 3634488961, 3889429448, 568446438, 3275163606, 4107603335, 1163531501,
                       2850285829, 4243563512, 1735328473, 2368359562, 4294588738, 2272392833, 1839030562,
                       4259657740, 2763975236, 1272893353, 4139469664, 3200236656, 681279174, 3936430074,
                       3572445317, 76029189, 3654602809, 3873151461, 530742520, 3299628645, 4096336452,
                       1126891415, 2878612391, 4237533241, 1700485571, 2399980690, 4293915773, 2240044497,
                       1873313359, 4264355552, 2734768916, 1309151649, 4149444226, 3174756917, 718787259, 3951481745};                       
  
  hipMemcpyToSymbol(HIP_SYMBOL(device_sines), host_sines, sizeof(host_sines));
*/
  // Copy target hash to the device (So that the comparison can be done on the GPU)
  hipMemcpyToSymbol(HIP_SYMBOL(deviceTarget), target, sizeof(deviceTarget));
  hipMemcpyToSymbol(HIP_SYMBOL(resultIndex), &nf, sizeof(nf));
  
}


// md5Hash(message, device, host, length);
bool doHash(std::vector<std::string>& keys) {
  using namespace std;

  // Getting the device properties.
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int numBlocks = deviceProp.multiProcessorCount * 2;
  int numThreadsPerBlock = NUM_THREADS_PER_BLOCK;
  int numThreadsPerGrid = numBlocks * numThreadsPerBlock;
  int sharedMem = deviceProp.sharedMemPerBlock / 2;

  // Array of pointers to each message on device memory.
  UCHAR* hostMsgLocationsOnDevice[numThreadsPerGrid];
  UCHAR** deviceMsgLocationsOnDevice;
  
  // Array of lengths of each message.
  int hostMsgLengths[numThreadsPerGrid];
  int* deviceMsgLengths;

  // Array of hash for each message.
  uint4 hostDigests[numThreadsPerGrid];
  uint4* deviceDigests;
  UCHAR result[56];

  hipError_t err = hipGetLastError();
  
// For each message
  for (int i = 0; i != keys.size(); ++i) {
    const char* key = keys[i].c_str();
    hostMsgLengths[i] = keys[i].size();
    
    // Create an array of pointers pointing to the hashes
    hipMalloc((void **)&hostMsgLocationsOnDevice[i], keys[i].length() * sizeof(UCHAR));
    hipMemcpy(hostMsgLocationsOnDevice[i], key, keys[i].length(), hipMemcpyHostToDevice);
    
  err = hipGetLastError();
  if (hipSuccess != err)
    printf("1: %s\n", hipGetErrorString(err));
    hipMalloc((void **)&deviceMsgLengths, numThreadsPerGrid * sizeof(int));
    hipMemcpy(deviceMsgLengths, hostMsgLengths, numThreadsPerGrid * sizeof(int), hipMemcpyHostToDevice);
  if (hipSuccess != err)
    printf("2: %s\n", hipGetErrorString(err));
  }
  
  hipMalloc((void **)&deviceMsgLocationsOnDevice, numThreadsPerGrid * sizeof(UCHAR*));
  hipMemcpy(deviceMsgLocationsOnDevice, hostMsgLocationsOnDevice, sizeof(hostMsgLocationsOnDevice), hipMemcpyHostToDevice);
  if (hipSuccess != err)
    printf("3: %s\n", hipGetErrorString(err));

  
  hipMalloc((void **)&deviceDigests, numThreadsPerGrid * sizeof(uint4));
  err = hipGetLastError();
  if (hipSuccess != err)
    printf("4: %s\n", hipGetErrorString(err));
    
  md5Hash <<< numBlocks, numThreadsPerBlock, sharedMem >>> (deviceMsgLocationsOnDevice, deviceMsgLengths, deviceDigests);
  //hipDeviceSynchronize();
  
    err = hipGetLastError();
  if (hipSuccess != err)
    printf("5: %s\n", hipGetErrorString(err));
  hipMemcpy(hostDigests, deviceDigests, sizeof(hostDigests), hipMemcpyDeviceToHost);
  int ri = NOT_FOUND;
  int* resultAddress;
  //  hipGetSymbolAddress((void**)&resultAddress, "resultIndex");
  
  
  if (hipSuccess == hipGetSymbolAddress((void**)&resultAddress, "resultIndex")) {
    //printf("OK\n");
  } else {
    printf("ERROR: Address of resultAddress is invalid!\n");
  };
  
  
  hipMemcpy(&ri, resultAddress, sizeof(int), hipMemcpyDeviceToHost);
  // Check if target hash was found
  if (ri >= 0){
    //printf("Result index: %d\n",ri);
    hipMemcpy(result, hostMsgLocationsOnDevice[ri], hostMsgLengths[ri], hipMemcpyDeviceToHost);
    printf("Hash found: ");
    //Prints 
    for(int i =0; i != hostMsgLengths[ri]; i++)
      putchar(result[i]);
    putchar('\n');
    
    return true;
    
    //printf("Data at result %08x %08x %08x %08x\n",result[0],result[1],result[2],result[3]);
  } else {
    return false;
    //printf("Target hash not found.\n");
  }
  err = hipGetLastError();
  if (hipSuccess != err)
    printf("9: %s\n", hipGetErrorString(err));

  // Free memory
  hipFree(hostMsgLocationsOnDevice);
  hipFree(deviceMsgLengths);  
}

__global__ void md5Hash(UCHAR** messages, int* msgLengths, uint4* digests) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  UINT* in = pad(messages[idx], msgLengths[idx]);
  
  //digests[idx] = make_uint4(paddedMessage[0], paddedMessage[1], paddedMessage[2], paddedMessage[14]);

  unsigned int h0 = 0x67452301;
  unsigned int h1 = 0xEFCDAB89;
  unsigned int h2 = 0x98BADCFE;
  unsigned int h3 = 0x10325476;
  
  UINT a = h0;
  UINT b = h1;
  UINT c = h2;
  UINT d = h3;
  
    /* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
  FF ( a, b, c, d, in[ 0], S11, 3614090360); /* 1 */
  FF ( d, a, b, c, in[ 1], S12, 3905402710); /* 2 */
  FF ( c, d, a, b, in[ 2], S13,  606105819); /* 3 */
  FF ( b, c, d, a, in[ 3], S14, 3250441966); /* 4 */
  FF ( a, b, c, d, in[ 4], S11, 4118548399); /* 5 */
  FF ( d, a, b, c, in[ 5], S12, 1200080426); /* 6 */
  FF ( c, d, a, b, in[ 6], S13, 2821735955); /* 7 */
  FF ( b, c, d, a, in[ 7], S14, 4249261313); /* 8 */
  FF ( a, b, c, d, in[ 8], S11, 1770035416); /* 9 */
  FF ( d, a, b, c, in[ 9], S12, 2336552879); /* 10 */
  FF ( c, d, a, b, in[10], S13, 4294925233); /* 11 */
  FF ( b, c, d, a, in[11], S14, 2304563134); /* 12 */
  FF ( a, b, c, d, in[12], S11, 1804603682); /* 13 */
  FF ( d, a, b, c, in[13], S12, 4254626195); /* 14 */
  FF ( c, d, a, b, in[14], S13, 2792965006); /* 15 */
  FF ( b, c, d, a, in[15], S14, 1236535329); /* 16 */
  
  /* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
  GG ( a, b, c, d, in[ 1], S21, 4129170786); /* 17 */
  GG ( d, a, b, c, in[ 6], S22, 3225465664); /* 18 */
  GG ( c, d, a, b, in[11], S23,  643717713); /* 19 */
  GG ( b, c, d, a, in[ 0], S24, 3921069994); /* 20 */
  GG ( a, b, c, d, in[ 5], S21, 3593408605); /* 21 */
  GG ( d, a, b, c, in[10], S22,   38016083); /* 22 */
  GG ( c, d, a, b, in[15], S23, 3634488961); /* 23 */
  GG ( b, c, d, a, in[ 4], S24, 3889429448); /* 24 */
  GG ( a, b, c, d, in[ 9], S21,  568446438); /* 25 */
  GG ( d, a, b, c, in[14], S22, 3275163606); /* 26 */
  GG ( c, d, a, b, in[ 3], S23, 4107603335); /* 27 */
  GG ( b, c, d, a, in[ 8], S24, 1163531501); /* 28 */
  GG ( a, b, c, d, in[13], S21, 2850285829); /* 29 */
  GG ( d, a, b, c, in[ 2], S22, 4243563512); /* 30 */
  GG ( c, d, a, b, in[ 7], S23, 1735328473); /* 31 */
  GG ( b, c, d, a, in[12], S24, 2368359562); /* 32 */

  /* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
  HH ( a, b, c, d, in[ 5], S31, 4294588738); /* 33 */
  HH ( d, a, b, c, in[ 8], S32, 2272392833); /* 34 */
  HH ( c, d, a, b, in[11], S33, 1839030562); /* 35 */
  HH ( b, c, d, a, in[14], S34, 4259657740); /* 36 */
  HH ( a, b, c, d, in[ 1], S31, 2763975236); /* 37 */
  HH ( d, a, b, c, in[ 4], S32, 1272893353); /* 38 */
  HH ( c, d, a, b, in[ 7], S33, 4139469664); /* 39 */
  HH ( b, c, d, a, in[10], S34, 3200236656); /* 40 */
  HH ( a, b, c, d, in[13], S31,  681279174); /* 41 */
  HH ( d, a, b, c, in[ 0], S32, 3936430074); /* 42 */
  HH ( c, d, a, b, in[ 3], S33, 3572445317); /* 43 */
  HH ( b, c, d, a, in[ 6], S34,   76029189); /* 44 */
  HH ( a, b, c, d, in[ 9], S31, 3654602809); /* 45 */
  HH ( d, a, b, c, in[12], S32, 3873151461); /* 46 */
  HH ( c, d, a, b, in[15], S33,  530742520); /* 47 */
  HH ( b, c, d, a, in[ 2], S34, 3299628645); /* 48 */
  
    /* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
  II ( a, b, c, d, in[ 0], S41, 4096336452); /* 49 */
  II ( d, a, b, c, in[ 7], S42, 1126891415); /* 50 */
  II ( c, d, a, b, in[14], S43, 2878612391); /* 51 */
  II ( b, c, d, a, in[ 5], S44, 4237533241); /* 52 */
  II ( a, b, c, d, in[12], S41, 1700485571); /* 53 */
  II ( d, a, b, c, in[ 3], S42, 2399980690); /* 54 */
  II ( c, d, a, b, in[10], S43, 4293915773); /* 55 */
  II ( b, c, d, a, in[ 1], S44, 2240044497); /* 56 */
  II ( a, b, c, d, in[ 8], S41, 1873313359); /* 57 */
  II ( d, a, b, c, in[15], S42, 4264355552); /* 58 */
  II ( c, d, a, b, in[ 6], S43, 2734768916); /* 59 */
  II ( b, c, d, a, in[13], S44, 1309151649); /* 60 */
  II ( a, b, c, d, in[ 4], S41, 4149444226); /* 61 */
  II ( d, a, b, c, in[11], S42, 3174756917); /* 62 */
  II ( c, d, a, b, in[ 2], S43,  718787259); /* 63 */
  II ( b, c, d, a, in[ 9], S44, 3951481745); /* 64 */
  
  a += h0;
  b += h1;
  c += h2;
  d += h3;
  
  // Check to see if this is the target hash!
  if (a == deviceTarget[0] && b == deviceTarget[1] && c == deviceTarget[2] && d == deviceTarget[3]){
     resultIndex = idx;
  } else {
    //resultIndex = -1;  // For some reason this does not work!? (Program wont find target hash)
  }
  digests[idx] = make_uint4(a, b, c, d);
  
}

__device__ UINT* pad(UCHAR* message, int msgLength) {
  //UCHAR* m;
  //UINT* paddedMessage = 0;
  //UINT* paddedMessage = (UINT*)array;
  //UCHAR* m = (UCHAR*)&paddedMessage[16];
  //UINT* paddedMessage = (UINT*)&array;
//  UCHAR* m = (UCHAR*)&array + (blockIdx.x * threadIdx.x) ;
  
  UCHAR m[56];
  
  for (int i = 0; i != 56; ++i)
    m[i] = 0x00;
  
  
  for (int i = 0; i != msgLength; ++i)
    m[i] = message[i];
  
  m[msgLength] = 0x80;
  
  UINT* paddedMessage = ((UINT*)&array) + (((blockIdx.x * blockDim.x + threadIdx.x) * 16));
  
  for (int i = 0; i != 14; ++i) {
      paddedMessage[i] = (UINT)m[i*4+3] << 24 |
        (UINT)m[i*4+2] << 16 |
        (UINT)m[i*4+1] << 8 |
        (UINT)m[i*4];
  }
  
  paddedMessage[14] = msgLength << 3;
  paddedMessage[15] = msgLength >> 29;

  return paddedMessage;
}

